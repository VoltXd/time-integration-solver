#include "hip/hip_runtime.h"
#include "CUDA_Euler.cuh"

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Toolbox.hpp"

__global__ void helloCuda()
{
   printf("GPU: Hello CUDA!\n");
}

void gpuHello()
{
    helloCuda<<< 1, 1 >>>();
}

__device__ double device_linearODEwCC(double input, const double* outputDerivativesVector, const double* coefficientVector, unsigned long equationOrder) 
{
    double result = input;
    for (int i = 0; i < equationOrder; i++)
        result += coefficientVector[i] * outputDerivativesVector[i];
    return result;
}

ODE* deviceFPtr_highestDerivative;

__global__ void kernel_EulerSolver(unsigned long numberOfSamples, double* outputDerivativesVector, double* coefficientsVector, double* uVector, double* yVector, unsigned long equationOrder, double samplePeriod)
{
    double highestDerivativeValue;
    unsigned long currentSampleIndex = 1;
    for (; currentSampleIndex < numberOfSamples; currentSampleIndex++)
    {
        highestDerivativeValue = device_linearODEwCC(uVector[currentSampleIndex - 1], outputDerivativesVector, coefficientsVector, equationOrder);
        outputDerivativesVector[equationOrder - 1] += samplePeriod * highestDerivativeValue;
        for (int i = equationOrder - 2; 0 <= i; i--)
        {
            outputDerivativesVector[i] += samplePeriod * outputDerivativesVector[i + 1];  
        }
        yVector[currentSampleIndex] = outputDerivativesVector[0];
    }
}

void euler_iterateAll_CUDA(std::vector<double>& outputDerivativesVector, const std::vector<double>& coefficientsVector, const std::vector<double>& uVector, std::vector<double>& yVector, double samplePeriod)
{
    // I won't use the fPtr for the highest derivative value in CUDA
    // I'll use a hardcoded function (linear ODE w/ CC) for the moment

    unsigned long numberOfSamples = yVector.size();
    unsigned long equationOrder = coefficientsVector.size();

    // Allocate GPU memory
    double* cuda_outputDerivativesVector;
    double* cuda_coefficientsVector;
    double* cuda_uVector;
    double* cuda_yVector;

    hipMalloc(&cuda_outputDerivativesVector, outputDerivativesVector.size() * sizeof(double));
    hipMalloc(&cuda_coefficientsVector, coefficientsVector.size() * sizeof(double));
    hipMalloc(&cuda_uVector, uVector.size() * sizeof(double));
    hipMalloc(&cuda_yVector, yVector.size() * sizeof(double));

    // Copy datas in the GPU's memory
    hipMemcpy(cuda_outputDerivativesVector, outputDerivativesVector.data(), outputDerivativesVector.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_coefficientsVector, coefficientsVector.data(), coefficientsVector.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_uVector, uVector.data(), uVector.size() * sizeof(double), hipMemcpyHostToDevice);

    // Run the kernel
    kernel_EulerSolver<<<1, 1>>>(numberOfSamples, cuda_outputDerivativesVector, cuda_coefficientsVector, cuda_uVector, cuda_yVector, equationOrder, samplePeriod);

    // Copy the result in the CPU memory
    hipMemcpy(yVector.data(), cuda_yVector, yVector.size() * sizeof(double), hipMemcpyDeviceToHost);

    // FREE GPU MEMORY!!!!! (IMPORTANT)
    hipFree(cuda_outputDerivativesVector);
    hipFree(cuda_coefficientsVector);
    hipFree(cuda_uVector);
    hipFree(cuda_yVector);
}